
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime_api.h>

__global__ void add_kernel(int *C, const int *A, const int*B, const int n) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        C[i] = A[i] + B[i];
    }
}

int main(int argc, char **argv) {
    const int N = 128;
    const int BLOCK_DIM = 4;
    const int GRID_DIM = 3;
    int *C = new int[N];
    int *A = new int[N];
    int *B = new int[N];
    for (int i = 0; i < N; i++) {
        A[i] = i;
        B[i] = i * i;
    }

    int *Ag, *Bg, *Cg;
    hipMalloc(&Ag, N * sizeof(int));
    hipMalloc(&Bg, N * sizeof(int));
    hipMalloc(&Cg, N * sizeof(int));
    hipMemcpy(Ag, A, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(Bg, B, N * sizeof(int), hipMemcpyHostToDevice);

    add_kernel<<<GRID_DIM, BLOCK_DIM>>>(Cg, Ag, Bg, N);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cout << "CUDA Error: " << hipGetErrorString(error);
        exit(1);
    }

    hipMemcpy(C, Cg, N * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        std::cout << C[i] << " ";
    }
    std::cout << std::endl;
    return 0;
}