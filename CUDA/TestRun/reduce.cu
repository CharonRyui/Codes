// reduce.cu
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>



__global__ void reductionKernel(float* input, float* output, int size) {
    extern __shared__ float sharedData[];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    sharedData[tid] = (index < size) ? input[index] : 0;
    // __syncthreads();

    // Perform reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sharedData[tid] += sharedData[tid + stride];
        }
        // __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) {
        output[blockIdx.x] = sharedData[0];
    }
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void reduction_gpu(float* input, float* output, int size) {
    int blockSize = 256; // Number of threads per block
    int gridSize = (size + blockSize - 1) / blockSize;

    float* d_input, * d_output;
    checkCudaError(hipMalloc(&d_input, size * sizeof(float)), "Failed to allocate device memory for input");
    checkCudaError(hipMalloc(&d_output, gridSize * sizeof(float)), "Failed to allocate device memory for output");

    checkCudaError(hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice), "Failed to copy data to device");

    // Launch the reduction kernel using cudaLaunchKernel
    clock_t start = clock();
    void* kernelArgs[] = { &d_input, &d_output, &size };
    // void* kernelArgs[] = { &d_input, &d_output, &size };
    checkCudaError(hipLaunchKernel(reinterpret_cast<const void*>((void*)reductionKernel), 
                                    dim3(gridSize), dim3(blockSize), 
                                    kernelArgs, blockSize * sizeof(float), 0), 
                                    "Failed to launch reduction kernel");

    clock_t end = clock();
    double duration_kernel = (double)(end - start) / (CLOCKS_PER_SEC);
    printf("reduction_kernel: %.9lf\n", duration_kernel);

    // Check for any errors during kernel execution
    checkCudaError(hipGetLastError(), "Kernel execution failed");
    // Wait for the kernel to complete 
    hipDeviceSynchronize();

    end = clock();
    duration_kernel = (double)(end - start) / (CLOCKS_PER_SEC);
    printf("reduction_kernel after sync: %.9lf\n", duration_kernel);  
    // Copy the partial results back to the host
    checkCudaError(hipMemcpy(output, d_output, gridSize * sizeof(float), hipMemcpyDeviceToHost), "Failed to copy data back to host");
    // Perform the final reduction on the host
    float finalSum = 0.0f;
    for (int i = 0; i < gridSize; ++i) {
        finalSum += output[i];
    }

    std::cout << "Total sum: " << finalSum << std::endl;

    hipFree(d_input);
    hipFree(d_output);
}

int main() {
    const int size = 1024 * 1024 * 16;
    float* input = new float[size];
    float* output = new float[(size + 255) / 256];

    // Initialize input data
    for (int i = 0; i < size; ++i) {
        input[i] = static_cast<float>(2);
    }

    reduction_gpu(input, output, size);

    delete[] input;
    delete[] output;

    return 0;
}